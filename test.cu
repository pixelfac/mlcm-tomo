#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(){
    int i = threadIdx.x;
    printf("Hello World from GPU! %i\n", i);
}

int main() {
    cuda_hello<<<2,5>>>(); 
    return 0;
}